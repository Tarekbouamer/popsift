#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include <iomanip>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <limits>

#include <stdlib.h>
#include <errno.h>
#include <hip/hip_math_constants.h>

#include "features.h"
#include "sift_extremum.h"
#include "common/assist.h"
#include "common/debug_macros.h"

using namespace std;

namespace popsift {

/*************************************************************
 * class Centroid
 *************************************************************/

template<int M> // M = 1 .. 7
class Centroid
{
    static const int range = ( 1 << (M-1) );
    static const int len   = ( 1 << ( 7 - (M-1) ) );

    float center[len];
    float variance[len];
    int   count;
    int   _memberCount;

public:
    void reset()
    {
        for( int d=0; d<len; d++ )
            variance[d] = 0;
        count = 0;
    }

    void resetMemberCount()
    {
        _memberCount = 0;
    }

    void incMemberCount()
    {
        _memberCount++;
    }

    int getMemberCount() const
    {
        return _memberCount;
    }

    float descSum( const Descriptor* data, int d )
    {
        float sum = 0.0f;
        for( int i=0; i<range; i++ )
            sum += data->features[d*range+i];
        return sum;
    }

    void addToCenter( const Descriptor* data )
    {
        for(int d=0; d<len; d++)
        {
            center[d] += descSum( data, d );
        }
        count += 1;
    }

    void normalizeCenter( )
    {
        if( count > 1 )
            for( int d=0; d<len; d++ )
                center[d] /= count;
        count = 0;
    }

    void addToVariance( const Descriptor* data )
    {
        for(int d=0; d<len; d++)
        {
            variance[d] += fabsf( center[d] - descSum( data, d ) );
        }
        count += 1;
    }

    void normalizeVariance( )
    {
        if( count > 1 )
            for( int d=0; d<len; d++ )
                variance[d] /= count;
        count = 0;
    }

    void addVariance( const Centroid& c )
    {
        for( int d=0; d<len; d++ )
            center[d] = c.center[d] + c.variance[d];
    }

    void subVariance( const Centroid& c )
    {
        for( int d=0; d<len; d++ )
            center[d] = c.center[d] - c.variance[d];
    }

    float l2dist( const Descriptor* desc )
    {
        float sum = 0.0f;
        for( int d=0; d<len; d++ )
        {
            sum += powf( center[d] - descSum( desc, d ), 2 );
        }
        sum = sqrtf( sum );
        return sum;
    }

    void printCenter( ostream& ostr ) const
    {
        for( int d=0; d<len; d++ )
        {
            ostr<< std::setprecision(2) << center[d] << " ";
        }
    }
};

/*************************************************************
 * class LindeBuzoGray
 *
 * Some sources claim that Linde-Buzo-Gray is a k-means algorithm.
 * That is not really the case. Centroids are not recomputed from
 * all nodes during refinement, but only internally refined.
 * That provides a more homogeneous sets but it does not create
 * centroids that for a Voronoi diagram.
 *
 * This class implements a true k-means algorithm as well, which
 * can be chosen by calling run(local=false) instead of run().
 *************************************************************/

template<int M>
class LindeBuzoGray
{
    const int                       _rounds;

    std::vector<Centroid<M> >       _centroids;

    const std::vector<Descriptor*>& _data;
    const int                       _len;
    std::vector<int>                _centerIdx;

public:
    LindeBuzoGray( const std::vector<Descriptor*>& descriptorList, int powerOf2 )
        : _rounds( powerOf2 )
        , _centroids( 1 << _rounds )
        , _data( descriptorList )
        , _len( descriptorList.size() )
        , _centerIdx( _len, 0 )
    {
        std::cout << "Creating Linde-Buzo-Gray with " << _len << " elements for " << _rounds << " rounds (" << _centroids.size() << " centroids)" << std::endl;
    }

    void run( bool local = true )
    {
        initCenterIdx();
        for( int r=0; r<_rounds; r++ )
        {
            computeCentroids();
            newCenters( r );
            if( local )
                findNewCentroidLocal( );
            else
                findNewCentroidGlobal( r+1 );

            // debugPrintCentroids( std::cout, r+1 );

            debugClosestPointCount( std::cout, r+1 );
        }
    }

    const Centroid<M>& getCentroid( int centroidIdx ) const
    {
        return _centroids[centroidIdx];
    }

    int getCenter( int descIdx ) const
    {
        return _centerIdx[descIdx];
    }

    void findBestMatches( const Descriptor& desc, int& idx1, float& val1, int& idx2, float& val2 )
    {
        idx1 = idx2 = -1;
        val1 = val2 = std::numeric_limits<float>::max();

        for( int i=0; i<(1<<_rounds); i++ )
        {
            float f = _centroids[i].l2dist( &desc );
            if( f < val1 )
            {
                val2 = val1;
                val1 = f;
                idx2 = idx1;
                idx1 = i;
            }
            else if( f < val2 )
            {
                val2 = f;
                idx2 = i;
            }
        }
    }

private:
    void initCenterIdx()
    {
        for(auto& c : _centerIdx) c = 0;
    }

    void computeCentroids( )
    {
        for(auto& c : _centroids) c.reset();

        for( int i=0; i<_len; i++)
        {
            int ctr = _centerIdx[i];
            _centroids[ctr].addToCenter( _data[i] );
        }

        for(auto& c : _centroids) c.normalizeCenter();

        for( int i=0; i<_len; i++)
        {
            int ctr = _centerIdx[i];
            _centroids[ctr].addToVariance( _data[i] );
        }

        for(auto& c : _centroids) c.normalizeVariance();
    }

    void newCenters( int round )
    {
        int last = 1 << round;

        for( int i=last-1; i>=0; i-- )
        {
            _centroids[i*2+1].addVariance( _centroids[i] );
            _centroids[i*2+0].subVariance( _centroids[i] );
        }
    }

    void findNewCentroidLocal( )
    {
        for( int i=0; i<_len; i++)
        {
            int ctr = _centerIdx[i];
            _centroids[ctr*2+0].resetMemberCount();
            _centroids[ctr*2+1].resetMemberCount();
            const float dist0 = _centroids[ctr*2+0].l2dist( _data[i] );
            const float dist1 = _centroids[ctr*2+1].l2dist( _data[i] );
            const int   centr = dist0 < dist1
                              ? ctr*2+0
                              : ctr*2+1;
            _centerIdx[i] = centr;
            _centroids[centr].incMemberCount();
        }
    }

    void findNewCentroidGlobal( int round )
    {
        int num = 1 << round;
        std::vector<float> dist( num );

        for( int c=0; c<num; c++ )
        {
            _centroids[c].resetMemberCount();
        }

        for( int i=0; i<_len; i++)
        {
            for( int c=0; c<num; c++ )
            {
                dist[c] = _centroids[c].l2dist( _data[i] );
            }
            const int centr = std::min_element( dist.begin(), dist.end() ) - dist.begin();
            _centerIdx[i] = centr;
            _centroids[centr].incMemberCount();
        }
    }

    void debugPrintCentroids( ostream& ostr, int round )
    {
        int num = 1 << round;
        for( int i=0; i<num; i++ )
        {
            ostr<< i << ": ";
            _centroids[i].printCenter( ostr );
            ostr<< std::endl;
        }
    }

    void debugClosestPointCount( ostream& ostr, int round )
    {
        int num = 1 << round;
        for( int i=0; i<num; i++ )
        {
            ostr<< i << ": " << std::count( _centerIdx.begin(), _centerIdx.end(), i ) << std::endl;
        }
    }
};

/*************************************************************
 * class PqtAnn
 * Inspired by the paper 10.1109/CVPR.2016.223
 *************************************************************/

class PqtAnn
{
    static const int level1_rounds   = 4;
    static const int level1_clusters = 1 << level1_rounds;
    static const int level2_rounds   = 5;
    static const int level2_clusters = 1 << level1_rounds;

    template<int M>
    struct Level
    {
        vector<Descriptor*> desc; // copied from caller
        LindeBuzoGray<M>*   lbg;

        Level( )
            : lbg( 0 )
        { }

        Level( const std::vector<Descriptor*>& descriptorList, int rounds )
            : desc( descriptorList )
        {
            lbg = new LindeBuzoGray<M>( desc, rounds );
        }

        ~Level( )
        {
            delete lbg;
        }

        void makeLbg( int rounds )
        {
            lbg = new LindeBuzoGray<M>( desc, rounds );
        }
    };

    Level<5>              _level1;
    std::vector<Level<1>> _level2;
public:
    PqtAnn( const std::vector<Descriptor*>& descriptorList )
        : _level1( descriptorList, level1_rounds )
    { }

    void run( )
    {
        _level1.lbg->run( );

        _level2.resize( level1_clusters );

        for( int lvl1ctr=0; lvl1ctr<level1_clusters; lvl1ctr++ )
        {
            const int memCt = _level1.lbg->getCentroid(lvl1ctr).getMemberCount();
            _level2[lvl1ctr].desc.reserve( memCt );
        }

        for( int descIdx=0; descIdx<_level1.desc.size(); descIdx++ )
        {
            int centroidIdx = _level1.lbg->getCenter( descIdx );
            _level2[centroidIdx].desc.push_back( _level1.desc[descIdx] );
        }

        for( int lvl1ctr=0; lvl1ctr<level1_clusters; lvl1ctr++ )
        {
            _level2[lvl1ctr].makeLbg( level2_rounds );
            _level2[lvl1ctr].lbg->run();
        }
    }

    void findMatch( const Descriptor& desc )
    {
        int   idx1, idx2;
        float val1, val2;

        _level1.lbg->findBestMatches( desc, idx1, val1, idx2, val2 );

        std::cout << "matches -"
                  << setprecision(3);

        if( idx1 >= 0 )
        {
            int   idx3, idx4;
            float val3, val4;
            _level2[idx1].lbg->findBestMatches( desc, idx3, val3, idx4, val4 );

            if( idx4 >= 0 )
            {
                std::cout << " 0: " << val3;
                std::cout << " 1: " << val4;
            }
            else if( idx3 >= 0 )
            {
                std::cout << " 0: " << val3;
            }
        }

        if( idx2 >= 0 )
        {
            int   idx3, idx4;
            float val3, val4;
            _level2[idx2].lbg->findBestMatches( desc, idx3, val3, idx4, val4 );

            if( idx4 >= 0 )
            {
                std::cout << " 2: " << val3;
                std::cout << " 3: " << val4;
            }
            else if( idx3 >= 0 )
            {
                std::cout << " 2: " << val3;
            }
        }
        std::cout << std::endl;
    }
};

/*************************************************************
 * FeaturesBase
 *************************************************************/

FeaturesBase::FeaturesBase( )
    : _num_ext( 0 )
    , _num_ori( 0 )
{ }

FeaturesBase::~FeaturesBase( )
{ }

/*************************************************************
 * FeaturesHost
 *************************************************************/

FeaturesHost::FeaturesHost( )
    : _ext( 0 )
    , _ori( 0 )
{ }

FeaturesHost::FeaturesHost( int num_ext, int num_ori )
    : _ext( 0 )
    , _ori( 0 )
{
    reset( num_ext, num_ori );
}

FeaturesHost::~FeaturesHost( )
{
    memalign_free( _ext );
    memalign_free( _ori );
}

void FeaturesHost::reset( int num_ext, int num_ori )
{
    if( _ext != 0 ) { free( _ext ); _ext = 0; }
    if( _ori != 0 ) { free( _ori ); _ori = 0; }

    _ext = (Feature*)memalign( getPageSize(), num_ext * sizeof(Feature) );
    if( _ext == 0 ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime error:" << endl
             << "    Failed to (re)allocate memory for downloading " << num_ext << " features" << endl;
        if( errno == EINVAL ) cerr << "    Alignment is not a power of two." << endl;
        if( errno == ENOMEM ) cerr << "    Not enough memory." << endl;
        exit( -1 );
    }
    _ori = (Descriptor*)memalign( getPageSize(), num_ori * sizeof(Descriptor) );
    if( _ori == 0 ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime error:" << endl
             << "    Failed to (re)allocate memory for downloading " << num_ori << " descriptors" << endl;
        if( errno == EINVAL ) cerr << "    Alignment is not a power of two." << endl;
        if( errno == ENOMEM ) cerr << "    Not enough memory." << endl;
        exit( -1 );
    }

    setFeatureCount( num_ext );
    setDescriptorCount( num_ori );
}

void FeaturesHost::pin( )
{
    hipError_t err;
    err = hipHostRegister( _ext, getFeatureCount() * sizeof(Feature), 0 );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime warning:" << endl
             << "    Failed to register feature memory in CUDA." << endl
             << "    Features count: " << getFeatureCount() << endl
             << "    Memory size requested: " << getFeatureCount() * sizeof(Feature) << endl
             << "    " << hipGetErrorString(err) << endl;
    }
    err = hipHostRegister( _ori, getDescriptorCount() * sizeof(Descriptor), 0 );
    if( err != hipSuccess ) {
        cerr << __FILE__ << ":" << __LINE__ << " Runtime warning:" << endl
             << "    Failed to register descriptor memory in CUDA." << endl
             << "    Descriptors count: " << getDescriptorCount() << endl
             << "    Memory size requested: " << getDescriptorCount() * sizeof(Descriptor) << endl
             << "    " << hipGetErrorString(err) << endl;
    }
}

void FeaturesHost::unpin( )
{
    hipHostUnregister( _ext );
    hipHostUnregister( _ori );
}

void FeaturesHost::print( std::ostream& ostr, bool write_as_uchar ) const
{
    for( int i=0; i<size(); i++ ) {
        _ext[i].print( ostr, write_as_uchar );
    }
}

void FeaturesHost::writeBinary( std::ostream& ostr, bool write_as_uchar ) const
{
    if( write_as_uchar )
    {
        ostr << "1 # writes descriptor as 128 uchars" << std::endl;
    }
    else
    {
        ostr << "0 # writes descriptor as 128 floats" << std::endl;
    }

    uint32_t num = getDescriptorCount();

    ostr << num << " # number of descriptors" << std::endl;

    for( int i=0; i<size(); i++ )
    {
        _ext[i].writeBinaryKeypoint( ostr );
    }
    int descriptors_written = 0;
    for( int i=0; i<size(); i++ )
    {
        descriptors_written += _ext[i].writeBinaryDescriptor( ostr, write_as_uchar );
    }
    std::cerr << "Written " << descriptors_written << " descriptors" << std::endl;
}

void FeaturesHost::debugCompareBinary( std::istream& verify, bool write_as_uchar ) const
{
    FeaturesHost dummy;
    dummy.readBinary( verify );
    if( getDescriptorCount() == dummy.getDescriptorCount() )
    {
        int read_ori_idx  = 0;
        int read_desc_idx = 0;

        for( int desc=0; desc<getDescriptorCount(); desc++ )
        {
            if( _ext[read_ori_idx].xpos != dummy._ext[desc].xpos )
            {
                std::cerr << "Written xpos bad on re-reading: "
                          << _ext[read_ori_idx].xpos << " vs " << dummy._ext[desc].xpos
                          << " for descriptor #" << desc
                          << std::endl;
                return;
            }

            if( _ext[read_ori_idx].ypos != dummy._ext[desc].ypos )
            {
                std::cerr << "Written ypos bad on re-reading: "
                          << _ext[read_ori_idx].ypos << " vs " << dummy._ext[desc].ypos
                          << " for descriptor #" << desc
                          << std::endl;
                return;
            }

            if( _ext[read_ori_idx].sigma != dummy._ext[desc].sigma )
            {
                std::cerr << "Written sigma bad on re-reading: "
                          << _ext[read_ori_idx].sigma << " vs " << dummy._ext[desc].sigma
                          << " for descriptor #" << desc
                          << std::endl;
                return;
            }

            if( _ext[read_ori_idx].orientation[read_desc_idx] != dummy._ext[desc].orientation[0] )
            {
                std::cerr << "Written orientation bad on re-reading: "
                          << _ext[read_ori_idx].orientation[read_desc_idx] << " vs " << dummy._ext[desc].orientation[0]
                          << " for descriptor #" << desc
                          << std::endl;
                return;
            }

            read_desc_idx++;
            if( read_desc_idx >= _ext[read_ori_idx].num_ori )
            {
                read_ori_idx++;
                read_desc_idx = 0;
            }
        }

        for( int desc=0; desc<getDescriptorCount(); desc++ )
        {
            for( int d=0; d<128; d++ )
            {
                float actual = _ori[desc].features[d];
                float reread = dummy._ori[desc].features[d];
                if( write_as_uchar )
                {
                    actual = (unsigned char)roundf(actual);
                }

                if( actual != reread )
                {
                    std::cerr << "Difference in descriptor " << desc << " dim " << d << ": "
                              << actual << " vs " << reread
                              << std::endl;
                    break;
                }
            }
        }
    }
    else
    {
        std::cerr << "Wrote " << getDescriptorCount() << " descriptors, reading " << dummy.getDescriptorCount() << std::endl;
    }
}

bool FeaturesHost::readBinary( std::istream& ostr )
{
    bool written_as_uchar = false;
    int  num_descriptors  = 0;

    {
        char buffer[1024];
        ostr.getline( buffer, 1024 );
        written_as_uchar = ( buffer[0] == '1' );
        ostr >> num_descriptors;
        ostr.getline( buffer, 1024 ); // read rest of line and discard
    }

    if( num_descriptors <= 0 )
    {
        return false;
    }

    reset( num_descriptors, num_descriptors ); // descriptors written multiple times

    for( int i=0; i<num_descriptors; i++ )
    {
        float desc[4];
        ostr.read( (char*)desc, 4*sizeof(float) );
        _ext[i].debug_octave   = 0;
        _ext[i].xpos           = desc[0];
        _ext[i].ypos           = desc[1];
        _ext[i].sigma          = desc[2];
        _ext[i].num_ori        = 1;
        _ext[i].orientation[0] = desc[3];
        _ext[i].desc[0]        = &_ori[i];
    }

    if( written_as_uchar )
    {
        std::cerr << "Reading " << num_descriptors << " uchar descriptors" << std::endl;
        unsigned char* v = new unsigned char[128 * num_descriptors];
        unsigned char* vit = v;
        ostr.read( (char*)v, 128 * num_descriptors * sizeof(unsigned char) );
        for( int i=0; i<num_descriptors; i++ )
        {
            for( int d=0; d<128; d++ )
            {
                _ori[i].features[d] = *vit;
                vit++;
            }
        }
        delete [] v;
    }
    else
    {
        std::cerr << "Reading " << num_descriptors << " float descriptors" << std::endl;

        // Descriptor contains only features[128], linear read should be equivalent to
        // for( int i=0; i<num_descriptors; i++ ) ostr.read( (char*)(_ori[i].features), 128 * sizeof(float) );
        ostr.read( (char*)(_ori[0].features), 128 * num_descriptors * sizeof(float) );
    }

    return true;
}

__global__
void fix_descriptor_pointers( Feature*    features,
                              int         feature_count,
                              Descriptor* old_base_ptr,
                              Descriptor* new_base_ptr )
{
    const int idx = blockIdx.x * 32 + threadIdx.x;
    if( idx > feature_count ) return;
    Feature& f = features[idx];
    for( int ori=0; ori<f.num_ori; ori++ )
    {
        f.desc[ori] = (Descriptor*)( (char*)(f.desc[ori]) - (char*)(old_base_ptr) + (char*)(new_base_ptr) );
    }
}

__global__
void fix_reverse_map( Feature*    features,
                      int         feature_count,
                      Descriptor* desc_base,
                      int*        reverseMap )
{
    const int idx = blockIdx.x * 32 + threadIdx.x;
    if( idx > feature_count ) return;
    Feature& f = features[idx];
    for( int ori=0; ori<f.num_ori; ori++ )
    {
        
        Descriptor* desc_this = f.desc[ori];
        int offset = desc_this - desc_base;
        reverseMap[offset] = idx;
    }
}

FeaturesDev* FeaturesHost::toDevice()
{
    FeaturesDev* dev_features = new FeaturesDev( getFeatureCount(), getDescriptorCount() );
    pin();
    popcuda_memcpy_sync( dev_features->getFeatures(),
                         getFeatures(),
                         getFeatureCount() * sizeof(Feature),
                         hipMemcpyHostToDevice );

    popcuda_memcpy_sync( dev_features->getDescriptors(),
                         getDescriptors(),
                         getDescriptorCount() * sizeof(Descriptor),
                         hipMemcpyHostToDevice );
    unpin();
    
    dim3 grid( grid_divide( getFeatureCount(), 32 ) );
    fix_descriptor_pointers
        <<<grid,32>>>
        ( dev_features->getFeatures(),
          getFeatureCount(),
          getDescriptors(),
          dev_features->getDescriptors() );
    fix_reverse_map
        <<<grid,32>>>
        ( dev_features->getFeatures(),
          dev_features->getFeatureCount(),
          dev_features->getDescriptors(),
          dev_features->getReverseMap() );
    return dev_features;
}

void FeaturesHost::match( FeaturesHost* other )
{
    int         l_len  = getDescriptorCount( );
    Descriptor* l_ori  = getDescriptors( );

    std::vector<Descriptor*> ori( l_len );
    for( int i=0; i<l_len; i++ )
        ori[i] = &l_ori[i];

    // const int   rounds = 4;
    // LindeBuzoGray<5> lbg( ori, rounds );
    // lbg.run( true );

    PqtAnn pqt( ori );
    pqt.run();

    int         r_len  = getDescriptorCount( );
    Descriptor* r_ori  = getDescriptors( );
    for( int i=0; i<r_len; i++ )
    {
        pqt.findMatch( r_ori[i] );
    }
}

std::ostream& operator<<( std::ostream& ostr, const FeaturesHost& feature )
{
    feature.print( ostr, false );
    return ostr;
}

/*************************************************************
 * FeaturesDev
 *************************************************************/

FeaturesDev::FeaturesDev( )
    : _ext( 0 )
    , _ori( 0 )
    , _rev( 0 )
{ }

FeaturesDev::FeaturesDev( int num_ext, int num_ori )
    : _ext( 0 )
    , _ori( 0 )
    , _rev( 0 )
{
    reset( num_ext, num_ori );
}

FeaturesDev::~FeaturesDev( )
{
    hipFree( _ext );
    hipFree( _ori );
    hipFree( _rev );
}

void FeaturesDev::reset( int num_ext, int num_ori )
{
    if( _ext != 0 ) { hipFree( _ext ); _ext = 0; }
    if( _ori != 0 ) { hipFree( _ori ); _ori = 0; }
    if( _rev != 0 ) { hipFree( _rev ); _rev = 0; }

    _ext = popsift::cuda::malloc_devT<Feature>   ( num_ext, __FILE__, __LINE__ );
    _ori = popsift::cuda::malloc_devT<Descriptor>( num_ori, __FILE__, __LINE__ );
    _rev = popsift::cuda::malloc_devT<int>       ( num_ori, __FILE__, __LINE__ );

    setFeatureCount( num_ext );
    setDescriptorCount( num_ori );
}

__device__ inline float
l2_in_t0( const float4* lptr, const float4* rptr )
{
    const float4  lval = lptr[threadIdx.x];
    const float4  rval = rptr[threadIdx.x];
    const float4  mval = make_float4( lval.x - rval.x,
			              lval.y - rval.y,
			              lval.z - rval.z,
			              lval.w - rval.w );
    float   res = mval.x * mval.x
	        + mval.y * mval.y
	        + mval.z * mval.z
	        + mval.w * mval.w;
    res += shuffle_down( res, 16 );
    res += shuffle_down( res,  8 );
    res += shuffle_down( res,  4 );
    res += shuffle_down( res,  2 );
    res += shuffle_down( res,  1 );
    return res;
}

__global__ void
compute_distance( int3* match_matrix, Descriptor* l, int l_len, Descriptor* r, int r_len )
{
    if( blockIdx.x >= l_len ) return;
    const int idx = blockIdx.x;

    float match_1st_val = HIP_INF_F;
    float match_2nd_val = HIP_INF_F;
    int   match_1st_idx = 0;
    int   match_2nd_idx = 0;

    const float4* lptr = (const float4*)( &l[idx] );

    for( int i=0; i<r_len; i++ )
    {
        const float4* rptr = (const float4*)( &r[i] );

        const float   res  = l2_in_t0( lptr, rptr );

        if( threadIdx.x == 0 )
        {
            if( res < match_1st_val )
            {
                match_2nd_val = match_1st_val;
                match_2nd_idx = match_1st_idx;
                match_1st_val = res;
                match_1st_idx = i;
            }
            else if( res < match_2nd_val )
            {
                match_2nd_val = res;
                match_2nd_idx = i;
            }
        }
        __syncthreads();
    }

    if( threadIdx.x == 0 )
    {
        bool accept = ( match_1st_val / match_2nd_val < 0.8f );
        match_matrix[blockIdx.x] = make_int3( match_1st_idx, match_2nd_idx, accept );
    }
}

__global__ void
show_distance( int3*       match_matrix,
               Feature*    l_ext,
               Descriptor* l_ori,
               int*        l_fem,
               int         l_len,
               Feature*    r_ext,
               Descriptor* r_ori,
               int*        r_fem,
               int         r_len )
{
    for( int i=0; i<l_len; i++ )
    {
        const float4* lptr  = (const float4*)( &l_ori[i] );
        const float4* rptr1 = (const float4*)( &r_ori[match_matrix[i].x] );
        const float4* rptr2 = (const float4*)( &r_ori[match_matrix[i].y] );
        float d1 = l2_in_t0( lptr, rptr1 );
        float d2 = l2_in_t0( lptr, rptr2 );
        if( threadIdx.x == 0 )
        {
            if( match_matrix[i].z )
                printf( "accept feat %4d [%4d] matches feat %4d [%4d] ( 2nd feat %4d [%4d] ) dist %.3f vs %.3f\n",
                        l_fem[i], i,
                        r_fem[match_matrix[i].x], match_matrix[i].x,
                        r_fem[match_matrix[i].y], match_matrix[i].y,
                        d1, d2 );
	    else
                printf( "reject feat %4d [%4d] matches feat %4d [%4d] ( 2nd feat %4d [%4d] ) dist %.3f vs %.3f\n",
                        l_fem[i], i,
                        r_fem[match_matrix[i].x], match_matrix[i].x,
                        r_fem[match_matrix[i].y], match_matrix[i].y,
                        d1, d2 );
        }
        __syncthreads();
    }
}

void FeaturesDev::match( FeaturesDev* other )
{
    int l_len = getDescriptorCount( );
    int r_len = other->getDescriptorCount( );

    int3* match_matrix = popsift::cuda::malloc_devT<int3>( l_len, __FILE__, __LINE__ );

    dim3 grid;
    grid.x = l_len;
    grid.y = 1;
    grid.z = 1;
    dim3 block;
    block.x = 32;
    block.y = 1;
    block.z = 1;

    compute_distance
        <<<grid,block>>>
        ( match_matrix, getDescriptors(), l_len, other->getDescriptors(), r_len );

    POP_SYNC_CHK;

    show_distance
        <<<1,32>>>
        ( match_matrix,
          getFeatures(),
          getDescriptors(),
          getReverseMap(),
          l_len,
          other->getFeatures(),
          other->getDescriptors(),
          other->getReverseMap(),
          r_len );

    POP_SYNC_CHK;

    hipFree( match_matrix );
}

/*************************************************************
 * Feature
 *************************************************************/

void Feature::print( std::ostream& ostr, bool write_as_uchar ) const
{
    float sigval =  1.0f / ( sigma * sigma );

    for( int ori=0; ori<num_ori; ori++ ) {
        ostr << xpos << " " << ypos << " "
             << sigval << " 0 " << sigval << " ";
        if( write_as_uchar ) {
            for( int i=0; i<128; i++ ) {
                ostr << roundf(desc[ori]->features[i]) << " ";
            }
        } else {
            ostr << std::setprecision(3);
            for( int i=0; i<128; i++ ) {
                ostr << desc[ori]->features[i] << " ";
            }
            ostr << std::setprecision(6);
        }
        ostr << std::endl;
    }
}

void Feature::writeBinaryKeypoint( std::ostream& ostr ) const
{
    float keypoint[4];
    keypoint[0] = xpos;
    keypoint[1] = ypos;
    keypoint[2] = sigma;

    for( int ori=0; ori<num_ori; ori++ )
    {
        keypoint[3] = orientation[ori];
        ostr.write( (const char*)keypoint, 4*sizeof(float) );
    }
}

int Feature::writeBinaryDescriptor( std::ostream& ostr, bool write_as_uchar ) const
{
    int descriptors_written = 0;
    for( int ori=0; ori<num_ori; ori++ )
    {
        if( write_as_uchar )
        {
            unsigned char buffer[128];
            for( int i=0; i<128; i++ )
            {
                buffer[i] = (unsigned char)( roundf(desc[ori]->features[i]) );
            }
            ostr.write( (const char*)buffer, 128 * sizeof(unsigned char) );
            descriptors_written++;
        }
        else
        {
            ostr.write( (const char*)(desc[ori]->features), 128 * sizeof(float) );
            descriptors_written++;
        }
    }
    return descriptors_written;
}

std::ostream& operator<<( std::ostream& ostr, const Feature& feature )
{
    feature.print( ostr, false );
    return ostr;
}

} // namespace popsift

